#include "hip/hip_runtime.h"
#include <iostream>
#include <png++/png.hpp>
#include <cmath>
#include <chrono>


typedef struct pixel {
    
    png::byte red;
    png::byte blue;
    png::byte green;
    
    void from_rgb_pixel(png::rgb_pixel other) {
        this->red = other.red;
        this->blue = other.blue;
        this->green = other.green;
    }
    
    auto to_rgb_pixel() -> png::rgb_pixel const {
        return png::rgb_pixel(this->red, this->blue, this->green);
    }
} pixel;


__global__
void image_to_greyscale(pixel *pixels, pixel *result, size_t size) {
    for (size_t index = 0; index < size; index++) {
        pixel pixel = pixels[index];
        auto value = (png::byte) (sqrtf(
                    pixel.red * pixel.red +
                    pixel.blue * pixel.blue +
                    pixel.green * pixel.green) / 3);
        result[index] = {value, value, value};
    }
}


__global__
void image_to_negative(pixel *pixels, pixel *result, size_t size) {
    for (size_t index = 0; index < size; index++) {
        pixel pixel = pixels[index];
        result[index] = {
            (png::byte)(255 - pixel.red),
            (png::byte)(255 - pixel.blue),
            (png::byte)(255 - pixel.green)
        };
    }
}


int main() {

    png::image<png::rgb_pixel> original("original_image.png");
    pixel *pixels;
    pixel *result;

    size_t width = original.get_width();
    size_t height = original.get_height();

    hipMallocManaged(&pixels, width * height * sizeof(pixel));
    hipMallocManaged(&result, width * height * sizeof(pixel));

    for(size_t column_t = 0; column_t < width; column_t++) {
        for (size_t row_t = 0; row_t < height; row_t++) {
            pixels[column_t * height + row_t].from_rgb_pixel(
                    original.get_pixel(column_t, row_t));
        }
    }  

    auto start = std::chrono::system_clock::now();

    //image_to_negative<<<1, 1>>>(pixels, result, width * height);
    image_to_greyscale<<<1, height>>>(pixels, result, width * height);
    hipDeviceSynchronize();
    
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> time = end - start;
    std::cout << "Task took: " << time.count() << std::endl;

    for(size_t column_t = 0; column_t < width; column_t++)
        for(size_t row_t = 0; row_t < height; row_t++)
            original.set_pixel(column_t, row_t, result[column_t * height + row_t]
                .to_rgb_pixel());

    original.write("greyscale.png");
    hipFree(pixels);
    hipFree(result);
    return 0;
}

